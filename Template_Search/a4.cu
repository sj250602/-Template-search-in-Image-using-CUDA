
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

__device__ float bi_linear(float* d_image,int row,int col, float x, float y, int v){
    int x1 = floor(x);
    int y1 = ceil(y);
    return d_image[y1*col*3+x1*3+v]*(x1+1-x)*abs(-y1+1+y)+d_image[y1*col*3+(x1+1)*3+v]*(x-x1)*abs(y1-1-y)+d_image[(y1-1)*col*3+x1*3+v]*(x1+1-x)*abs(y-y1)+d_image[(y1-1)*col*3+(x1+1)*3+v]*(x-x1)*abs(y-y1);
}

__device__ float rotation_x(float x, float y, int angle){
    if(angle==45){
        return (x-y)/sqrt(2.0);
    }else{
        return (x+y)/sqrt(2.0);
    }
}

__device__ float rotation_y(float x, float y, int angle){
    if(angle==45){
        return (x+y)/sqrt(2.0);
    }else{
        return (y-x)/sqrt(2.0);
    }
}

__device__ float rmsd(int bit,int q_row,int q_col,float*d_image,float*q_image,int r,int c,int d_row,int d_col){
    if(bit==0){
        float res =0;
        for(int i=r;i>r-q_row;i--){
            for(int j=c;j<c+q_col;j++){
                for(int k=0;k<3;k++){
                    res+=pow(d_image[3*d_col*(i)+3*(j)+k]-q_image[3*q_col*(q_row-(r-i)-1)+ 3*(j-c)+k],2);
                }
            }
        }

        res/=(q_row*q_col*3);
        return sqrt(res);
    }
    else if(bit==1){
        float res =0;
        for(int i=r;i>r-q_row;i--){
            for(int j=c;j<c+q_col;j++){
                float x = c+rotation_x(j-c,r-i,45);
                float y = r-rotation_y(j-c,r-i,45);
                for(int k=0;k<3;k++){
                    float red = bi_linear(d_image,d_row,d_col,x,y,k);
                    res+=pow(red-q_image[3*q_col*(q_row-1-(r-i))+ 3*(j-c) +k],2);
                }
            }
        }
        res/=(q_row*q_col*3);
        return sqrt(res);
    }else if(bit==2){
        float res =0;
        for(int i=r;i>r-q_row;i--){
            for(int j=c;j<c+q_col;j++){
                float x = c+rotation_x(j-c,r-i,-45);
                float y = r-rotation_y(j-c,r-i,-45);
                for(int k=0;k<3;k++){
                    float red = bi_linear(d_image,d_row,d_col,x,y,k);
                    res+=pow(red-q_image[3*q_col*(q_row-1-(r-i))+ 3*(j-c) +k],2);
                }
            }
        }
        res/=(q_row*q_col*3);
        return sqrt(res);
    }
    return 0;
}

__global__ void start_comp(float* d_image,float* q_image,int d_row,int d_col,int q_row,int q_col,float q_avg,float th2,float th1,float* res){

    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int pixel = 3*(bid*128+tid);
    int d_row_num = pixel/(3*d_col);
    int d_col_num = (pixel/3)%(d_col);
    if(pixel<d_row*d_col*3){
        //for zero
        if(d_row_num-q_row>=-1 && d_col_num+q_col<=d_col){
            float d_avg=0.0;
            for(int i=d_row_num;i>d_row_num-q_row;i--){
                for(int j=d_col_num;j<(d_col_num+q_col);j++){
                    d_avg+=((d_image[d_col*i*3+j*3]+d_image[d_col*i*3+j*3+1]+d_image[d_col*i*3+j*3+2])/3);
                }
            }
            d_avg/=(q_row*q_col);
            if(abs(d_avg-q_avg)<th2){
                
                float d_rmsd0 = rmsd(0,q_row,q_col,d_image,q_image,d_row_num,d_col_num,d_row,d_col);
                if(d_rmsd0<th1){
                    res[3*d_row_num*d_col+d_col_num*3] = d_rmsd0;
                }
            }
        }

        //for 45 
        float B_x = d_col_num+rotation_x(q_col-1,0,45);
        float B_y = d_row-1-d_row_num+rotation_y(q_col-1,0,45);


        float C_x = d_col_num+rotation_x(q_col-1,q_row-1,45);
        float C_y = d_row-1-d_row_num+rotation_y(q_col-1,q_row-1,45);


        float D_x = d_col_num+rotation_x(0,q_row-1,45);
        float D_y = d_row-1-d_row_num+rotation_y(0,q_row-1,45);

        int x1 = floor(D_x);
        int x2 = ceil(B_x);
        int y2 = ceil(C_y);
        
        if(x1>=0&& x2<d_col&&y2-d_row+1<=0&&d_row_num>=0){
            float d_avg = 0.0;
            for(int r=d_row_num;r>=d_row-1-y2;r--){
                for(int c=x1;c<=x2;c++){
                    d_avg+=((d_image[d_col*r*3+c*3]+d_image[d_col*r*3+c*3+  1]+d_image[d_col*r*3+c*3+2])/3);
                }
            }
            d_avg/=abs(d_row_num-(d_row-1-y2)+1)*abs(x2-x1+1);

            if(abs(d_avg-q_avg)<th2){
                float d_rmsd1 = rmsd(1,q_row,q_col,d_image,q_image,d_row_num,d_col_num,d_row,d_col);
                if(d_rmsd1<th1){
                    
                    res[3*d_row_num*d_col+d_col_num*3+1] = d_rmsd1;
                    
                }
            }
        }
        // //-45

        float b_x = d_col_num+rotation_x(q_col-1,0,-45);
        float b_y = d_row-1-d_row_num+rotation_y(q_col-1,0,-45);


        float c_x = d_col_num+rotation_x(q_col-1,q_row-1,-45);
        float c_y = d_row-1-d_row_num+rotation_y(q_col-1,q_row-1,-45);


        float d_x = d_col_num+rotation_x(0,q_row-1,-45);
        float d_y = d_row-1-d_row_num+rotation_y(0,q_row-1,-45);

        int y11 = ceil(d_y);
        int x22 = ceil(c_x);
        int y22 = floor(b_y);

        if(y22>=0&& x22<d_col&&y11-d_row+1<=0&&d_col_num>=0){
            float d_avg = 0.0;
            for(int r=d_row-1-y22;r>=d_row-1-y11;r--){
                for(int c=d_col_num;c<=x22;c++){
                    d_avg+=((d_image[d_col*r*3+c*3]+d_image[d_col*r*3+c*3+  1]+d_image[d_col*r*3+c*3+2])/3);
                }
            }
            d_avg/=abs(y11-y22+1)*abs(x22-d_col_num+1);
            float abs_d_avg = abs(d_avg-q_avg);

            if(abs_d_avg<th2){
                float d_rmsd2 = rmsd(2,q_row,q_col,d_image,q_image,d_row_num,d_col_num,d_row,d_col);

                if(d_rmsd2<th1){
                    
                    res[3*d_row_num*d_col+d_col_num*3+2] = d_rmsd2;
                    
                }
            }
        }
    }
}

class ans_tuple{
    public:
        int row,col,bit;
        float dis;
    ans_tuple(){
    }

    ans_tuple(float a,int b,int c,int d){
        dis = a;
        row = b;
        col = c;
        bit = d;
    }
};

struct grtr_tuple{
    bool operator()(const ans_tuple &t1,const ans_tuple &t2){
        return t1.dis<t2.dis;
    }
};

int main(int argc, char* argv[])
{
  ios_base::sync_with_stdio(false);
  cin.tie(NULL);cout.tie(NULL);
  assert (argc > 5);
  string d_str = argv[1];
  string q_str = argv[2];
  float th1 = stof(argv[3]);
  float th2 = stof(argv[4]);
  int n = stoi(argv[5]);

  int d_row, d_col, q_row, q_col,k=0;
  float *d_image,*q_image;
  float q_avg;
  ifstream d_file(d_str),q_file(q_str);
  string str;
  getline(d_file,str);
  stringstream ss(str);
  string word;
  ss>>word;
  d_row = stoi(word);
  ss>>word;
  d_col = stoi(word);
  d_image = new float[d_row*d_col*3];


  getline(d_file,str);
  d_file.close();
  stringstream s(str);
  for(int i=0;i<d_row;i++){
      for(int j=0;j<d_col;j++){
          float a,b,c;
          s>>word;
          a = stof(word);
          s>>word;
          b = stof(word);
          s>>word;
          c = stof(word);
          d_image[k++] = a;
          d_image[k++] = b;
          d_image[k++] = c;
      }
  }

  k=0;
  getline(q_file,str);
  stringstream ss1(str);
  ss1>>word;
  q_row = stoi(word);
  ss1>>word;
  q_col = stoi(word);
  q_image = new float[q_row*q_col*3];
  getline(q_file,str);
  q_file.close();
  stringstream s1(str);
  for(int i=0;i<q_row;i++){
      vector<vector<float>> vect;
      vector<float> grey;
      for(int j=0;j<q_col;j++){
          float a,b,c;
          s1>>word;
          a = stof(word);
          s1>>word;
          b = stof(word);
          s1>>word;
          c = stof(word);
          q_image[k++] = a;
          q_image[k++] = b;
          q_image[k++] = c;
          q_avg += ((a+b+c)/3);
      }
  }
    
  q_avg/=(q_row*q_col);
  
  int d_isc = 3*d_row*d_col;
  int d_gsc = d_row*d_col;
  int q_isc = 3*q_row*q_col;
  float* o_d_image,*o_q_image,*res;
  float* res_c = new float[d_isc];

  hipMalloc(&o_d_image,d_isc*sizeof(float));
  hipMalloc(&o_q_image,q_isc*sizeof(float));
  hipMalloc(&res,d_isc*sizeof(float));

  for(int i=0;i<d_isc;i++){
      res_c[i] = -100000.0;
  }
  
  hipMemcpy(o_d_image,d_image,d_isc*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(o_q_image,q_image,q_isc*sizeof(float),hipMemcpyHostToDevice);
  hipMemcpy(res,res_c,d_isc*sizeof(float),hipMemcpyHostToDevice);  

  start_comp<<<(d_gsc+127)/128,128>>>(o_d_image,o_q_image,d_row,d_col,q_row,q_col,q_avg,th2,th1,res);
  hipMemcpy(res_c,res,d_isc*sizeof(float),hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  priority_queue <ans_tuple,vector<ans_tuple>,grtr_tuple> pq;
  for(int i=0;i<d_row;i++){
    for(int j=0;j<d_col;j++){
        for(int k=0;k<3;k++){
            float rmsd_error = res_c[3*i*d_col+3*j+k];
            if(rmsd_error!=-100000){
                pq.push(ans_tuple(rmsd_error,d_row-1-i,j,k));
                if(pq.size()>n){
                    pq.pop();
                }
            }
        }
    }
  }

  vector<ans_tuple> ans(n);
  int len = n-1;
  while(!pq.empty()){
      ans[len--] = pq.top();
      if(ans[len+1].bit==1){
        ans[len+1].bit = 45;
      }else if(ans[len+1].bit==2){
        ans[len+1].bit = -45;
      }
      pq.pop();
  }

  hipFree(o_d_image);
  hipFree(o_q_image);
  hipFree(res);
  free(d_image);
  free(q_image);

  ofstream output("output.txt");
  for(int i=0;i<n;i++){
    output<<ans[i].row<<" "<<ans[i].col<<" "<<ans[i].bit<<"\n";
  }
  output.close();


}
